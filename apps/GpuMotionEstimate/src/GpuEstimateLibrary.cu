#include "hip/hip_runtime.h"
// Copyright 2017 Caoyang Jiang

#include "GpuEstimateLibrary.cuh"

#include <cstdint>
#include <iostream>
#include <vector>

__global__ static void CUDAInterEstimateKernel(
    int16_t* in, int16_t* ref, size_t count, uint8_t* mv, int16_t* res)
{
  int id          = threadIdx.x + blockIdx.x * blockDim.x;
  uint16_t sad[4] = {65535, 65535, 65535, 65535};
  int16_t pin;
  uint8_t mvloc = 0;

  if (id < count)
  {
    pin = in[id];

    sad[0]                    = abs(pin - ref[id]);
    if ((id - 1) >= 0) sad[1] = abs(pin - ref[id - 1]);
    if ((id - 2) >= 0) sad[2] = abs(pin - ref[id - 2]);
    if ((id - 3) >= 0) sad[3] = abs(pin - ref[id - 3]);

    for (uint8_t i = 1; i < 4; i++)
    {
      if (sad[i] < sad[0])
      {
        sad[0] = sad[i];
        mvloc  = i;
      }
    }

    mv[id]  = mvloc;
    res[id] = pin - ref[id - mvloc];
  }
}

namespace jcy
{
void GpuPredictor::InterEstimate(const std::vector<int16_t>& in,
                                 const std::vector<int16_t>& ref,
                                 std::vector<uint8_t>& mv,
                                 std::vector<int16_t>& res)
{
  size_t count  = in.size();
  int thdinblk  = 512;
  int blkingrid = (count + thdinblk - 1) / thdinblk;
  int16_t* din;
  int16_t* dref;
  uint8_t* dmv;
  int16_t* dres;

  hipMalloc(reinterpret_cast<void**>(&din), count * sizeof(int16_t));
  hipMalloc(reinterpret_cast<void**>(&dref), count * sizeof(int16_t));
  hipMalloc(reinterpret_cast<void**>(&dmv), count * sizeof(uint8_t));
  hipMalloc(reinterpret_cast<void**>(&dres), count * sizeof(int16_t));

  dim3 blkdim(thdinblk, 1);
  dim3 griddim(blkingrid, 1);

  hipMemcpy(reinterpret_cast<void*>(din),
             reinterpret_cast<const void*>(in.data()),
             count * sizeof(int16_t),
             hipMemcpyHostToDevice);
  hipMemcpy(reinterpret_cast<void*>(dref),
             reinterpret_cast<const void*>(ref.data()),
             count * sizeof(int16_t),
             hipMemcpyHostToDevice);

  CUDAInterEstimateKernel<<<griddim, blkdim>>>(din, dref, count, dmv, dres);

  mv.resize(count);
  res.resize(count);
  hipMemcpy(reinterpret_cast<void*>(mv.data()),
             reinterpret_cast<const void*>(dmv),
             count * sizeof(uint8_t),
             hipMemcpyDeviceToHost);
  hipMemcpy(reinterpret_cast<void*>(res.data()),
             reinterpret_cast<const void*>(dres),
             count * sizeof(int16_t),
             hipMemcpyDeviceToHost);

  hipFree(din);
  hipFree(dmv);
  hipFree(dres);
  hipFree(dref);
}

void CpuPredictor::InterEstimate(const std::vector<int16_t>& in,
                                 const std::vector<int16_t>& ref,
                                 std::vector<uint8_t>& mv,
                                 std::vector<int16_t>& res)
{
  int count = static_cast<int>(in.size());
  mv.resize(count);
  res.resize(count);

  for (int c = 0; c < count; c++)
  {
    uint16_t sad[4] = {65535, 65535, 65535, 65535};
    int16_t pin;
    uint8_t mvloc = 0;

    pin = in[c];

    sad[0]                   = abs(pin - ref[c]);
    if ((c - 1) >= 0) sad[1] = abs(pin - ref[c - 1]);
    if ((c - 2) >= 0) sad[2] = abs(pin - ref[c - 2]);
    if ((c - 3) >= 0) sad[3] = abs(pin - ref[c - 3]);

    for (uint8_t i = 1; i < 4; i++)
    {
      if (sad[i] < sad[0])
      {
        sad[0] = sad[i];
        mvloc  = i;
      }
    }

    mv[c]  = mvloc;
    res[c] = pin - ref[c - mvloc];
  }
}
}  // namespace jcy

#include "hip/hip_runtime.h"
// Copyright 2015 Jason Juang

#include "Hvr/CUDASample/CUDASample.cuh"

HVR_WINDOWS_DISABLE_ALL_WARNING
#include "opencv2/opencv.hpp"
HVR_WINDOWS_ENABLE_ALL_WARNING

#include "Hvr/CUDASample/CUDAConfig.h"

__global__ static void CUDAKernelSetImageToWhite(int *data)
{
  const int x  = blockIdx.x * blockDim.x + threadIdx.x;
  const int y  = blockIdx.y * blockDim.y + threadIdx.y;
  const int mx = gridDim.x * blockDim.x;

  data[y * mx + x] = 255;
}

namespace hvr
{
CUDASample::CUDASample()
{
}

CUDASample::~CUDASample()
{
}

void CUDASample::SetImageToWhite(cv::Mat &img) const
{
  if (img.empty()) return;

  const int h = img.rows;
  const int w = img.cols;

  CUDAConfig cudaconfig;
  cudaconfig.h_a = cudaconfig.align(h, cudaconfig.blk_h);
  cudaconfig.w_a = cudaconfig.align(w, cudaconfig.blk_w);

  const int h_a = cudaconfig.h_a;
  const int w_a = cudaconfig.w_a;

  int *data;
  hipMallocManaged(reinterpret_cast<void **>(&data),
                    cudaconfig.h_a * cudaconfig.w_a * sizeof(int),
                    hipMemAttachGlobal);

  for (int i = 0; i < h; i++)
    for (int j = 0; j < w; j++)
    {
      data[i * w_a + j] = img.at<uchar>(i, j);
    }

  dim3 blks((w_a >> cudaconfig.shift_w), (h_a >> cudaconfig.shift_h));
  dim3 threads(cudaconfig.blk_w, cudaconfig.blk_h);

  CUDAKernelSetImageToWhite<<<blks, threads>>>(data);

  hipDeviceSynchronize();

  for (int i = 0; i < h; i++)
    for (int j = 0; j < w; j++)
    {
      img.at<uchar>(i, j) = data[i * w_a + j];
    }

  hipFree(data);
}

}  // namespace hvr

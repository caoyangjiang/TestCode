#include "hip/hip_runtime.h"
// Copyright 2016 Caoyang Jiang

#include <hip/hip_runtime.h>
#include <cstdio>
#include "TestCode/HPC/CudaIntegration.h"

__device__ static double XSQUARE(double x)
{
  // return x * x;
  // return x * x + 2 * x;
  return sin(x);
  // return pow(x, 2);
}

__global__ static void IntegrationKernal(double startx,
                                         double increment,
                                         double* psum)
{
  const int x  = blockIdx.x * blockDim.x + threadIdx.x;
  const int y  = blockIdx.y * blockDim.y + threadIdx.y;
  const int mx = gridDim.x * blockDim.x;

  // for (size_t i      = 0; i < 100000; i++)
  psum[y * mx + x] = (XSQUARE(startx + (y * mx + x) * increment) +
                      XSQUARE(startx + (y * mx + x + 1) * increment)) *
                     increment / 2;
  // std::printf("%d %1.12f\n", y * mx + x, psum[y * mx + x]);
}

namespace TestCode
{
CudaIntegration::CudaIntegration()
{
}

CudaIntegration::~CudaIntegration()
{
}

bool CudaIntegration::StartIntegration(double startx,
                                       double endx,
                                       size_t tcount,
                                       std::vector<double>& area)
{
  double increment = (endx - startx) / tcount;
  double* d_area;

  hipMalloc(reinterpret_cast<void**>(&d_area), sizeof(double) * tcount);

  dim3 blks(tcount / 256, 1);
  dim3 threads(256, 1);
  IntegrationKernal<<<blks, threads>>>(startx, increment, d_area);
  hipDeviceSynchronize();

  area.assign(tcount, 0);
  hipMemcpy(
      area.data(), d_area, sizeof(double) * tcount, hipMemcpyDeviceToHost);
  hipFree(d_area);

  return true;
}

}  // namespace TestCode
